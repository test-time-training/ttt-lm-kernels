#include "hip/hip_runtime.h"
#include "mma.cuh"

#ifdef TEST_GROUP_WGMMA_MMA

struct test_mma_AB {
    template<int H, int W, int NW, typename K, kittens::ducks::wgmma::normal L1, kittens::ducks::wgmma::transposed L2>
    using valid = std::bool_constant<NW == 4 && H==4 && (2*W*H+W*K::value+H*K::value)<=256 && (W <= 4 || W == 8)>;
    static inline const std::string test_identifier = "wgmma_mma_AB";
    template<int H, int W, int NW, typename _K, kittens::ducks::wgmma::normal L1, kittens::ducks::wgmma::transposed L2>
     __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        constexpr int K = _K::value;
        for(int i = 0; i < H*16; i++) {
            for(int j = 0; j < W*16; j++) {
                float sum = 0;
                for(int k = 0; k < K*16; k++) {
                    sum += i_ref[i*16*K + k]*i_ref[(256*H*K) + k*16*W + j];
                }
                o_ref[i*16*W + j] = sum;
            }
        }
    }
    template<int H, int W, int NW, typename _K, kittens::ducks::wgmma::normal L1, kittens::ducks::wgmma::transposed L2>
    __device__ static void device_func(const kittens::bf16 *input, kittens::bf16 *output) {
        constexpr int K = _K::value;
        extern __shared__ kittens::alignment_dummy __shm[]; // this is the CUDA shared memory
        kittens::shared_allocator<1024> al((int*)&__shm[0]); 
        kittens::st_bf<H, K, L1> &a = al.allocate<kittens::st_bf<H, K, L1>>();
        kittens::st_bf<K, W, L2> &b = al.allocate<kittens::st_bf<K, W, L2>>();
        kittens::rt_fl<1, W> c;
        __shared__ cuda::barrier<cuda::thread_scope::thread_scope_block> barrier;
        if (threadIdx.x == 0) {init(&barrier, kittens::WARPGROUP_THREADS);}
        __syncthreads();
        kittens::warpgroup::load_async(a, input, K*16, barrier);
        kittens::warpgroup::load_async(b, input+a.num_elements, W*16, barrier);
        barrier.arrive_and_wait();
        kittens::warpgroup::mma_fence(c);
        kittens::warpgroup::mm_AB(c, a, b);
        kittens::warpgroup::mma_commit_group();
        kittens::warpgroup::mma_async_wait();
        kittens::warpgroup::store(output, c, W*16);
    }
};
struct test_mma_ABt {
    template<int H, int W, int NW, typename K, kittens::ducks::wgmma::normal L1, kittens::ducks::wgmma::normal L2>
    using valid = std::bool_constant<NW == 4 && H==4 && (2*W*H+W*K::value+H*K::value)<=256 && (W <= 4 || W == 8)>; // this is warp-level
    static inline const std::string test_identifier = "wgmma_mma_ABt";
    template<int H, int W, int NW, typename _K, kittens::ducks::wgmma::normal L1, kittens::ducks::wgmma::normal L2>
    __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        constexpr int K = _K::value;
        for(int i = 0; i < H*16; i++) {
            for(int j = 0; j < W*16; j++) {
                float sum = 0;
                for(int k = 0; k < K*16; k++) {
                    sum += i_ref[i*K*16+k]*i_ref[256*K*H + j*K*16+k];
                }
                o_ref[i*W*16+j] = sum;
            }
        }
    }
    template<int H, int W, int NW, typename _K, kittens::ducks::wgmma::normal L1, kittens::ducks::wgmma::normal L2>
    __device__ static void device_func(const kittens::bf16 *input, kittens::bf16 *output) {
        constexpr int K = _K::value;
        extern __shared__ kittens::alignment_dummy __shm[]; // this is the CUDA shared memory
        kittens::shared_allocator<1024> al((int*)&__shm[0]); 
        kittens::st_bf<H, K, L1> &a = al.allocate<kittens::st_bf<H, K, L1>>();
        kittens::st_bf<W, K, L2> &b = al.allocate<kittens::st_bf<W, K, L2>>();
        kittens::rt_fl<1, W> c;
        __shared__ cuda::barrier<cuda::thread_scope::thread_scope_block> barrier;
        if (threadIdx.x == 0) {init(&barrier, kittens::WARPGROUP_THREADS);}
        __syncthreads();
        kittens::warpgroup::load_async(a, input, K*16, barrier);
        kittens::warpgroup::load_async(b, input+a.num_elements, K*16, barrier);
        barrier.arrive_and_wait();
        kittens::warpgroup::mma_fence(c);
        kittens::warpgroup::mm_ABt(c, a, b);
        kittens::warpgroup::mma_commit_group();
        kittens::warpgroup::mma_async_wait();
        kittens::warpgroup::store(output, c, W*16);
    }
};
struct test_mma_AtB {
    template<int H, int W, int NW, typename K, kittens::ducks::wgmma::normal L1, kittens::ducks::wgmma::normal L2>
    using valid = std::bool_constant<NW == 4 && H==4 && (2*W*H+W*K::value+H*K::value)<=256 && (W <= 4 || W == 8)>; // this is warp-level
    static inline const std::string test_identifier = "wgmma_mma_AtB";
    template<int H, int W, int NW, typename _K, kittens::ducks::wgmma::normal L1, kittens::ducks::wgmma::normal L2>
     __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        constexpr int K = _K::value;
        for(int i = 0; i < H*16; i++) {
            for(int j = 0; j < W*16; j++) {
                float sum = 0;
                for(int k = 0; k < K*16; k++) {
                    sum += i_ref[k*16*H + i]*i_ref[(256*H*K) + k*16*W + j];
                }
                o_ref[i*16*W + j] = sum;
            }
        }
    }
    template<int H, int W, int NW, typename _K, kittens::ducks::wgmma::normal L1, kittens::ducks::wgmma::normal L2>
    __device__ static void device_func(const kittens::bf16 *input, kittens::bf16 *output) {
        constexpr int K = _K::value;
        extern __shared__ kittens::alignment_dummy __shm[]; // this is the CUDA shared memory
        kittens::shared_allocator<1024> al((int*)&__shm[0]); 
        kittens::st_bf<K, H, L1> &a = al.allocate<kittens::st_bf<K, H, L1>>();
        kittens::st_bf<K, W, L2> &b = al.allocate<kittens::st_bf<K, W, L2>>();
        kittens::rt_fl<1, W> c;
        __shared__ cuda::barrier<cuda::thread_scope::thread_scope_block> barrier;
        if (threadIdx.x == 0) {init(&barrier, kittens::WARPGROUP_THREADS);}
        __syncthreads();
        kittens::warpgroup::load_async(a, input, H*16, barrier);
        kittens::warpgroup::load_async(b, input+a.num_elements, W*16, barrier);
        barrier.arrive_and_wait();
        kittens::warpgroup::mma_fence(c);
        kittens::warpgroup::mm_AtB(c, a, b);
        kittens::warpgroup::mma_commit_group();
        kittens::warpgroup::mma_async_wait();
        kittens::warpgroup::store(output, c, W*16);
    }
};
struct test_mma_AtBt {
    template<int H, int W, int NW, typename K, kittens::ducks::wgmma::normal L1, kittens::ducks::wgmma::normal L2>
    using valid = std::bool_constant<NW == 4 && H==4 && (2*W*H+W*K::value+H*K::value)<=256 && (W <= 4 || W == 8)>; // this is warp-level
    static inline const std::string test_identifier = "wgmma_mma_AtBt";
    template<int H, int W, int NW, typename _K, kittens::ducks::wgmma::normal L1, kittens::ducks::wgmma::normal L2>
    __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        constexpr int K = _K::value;
        for(int i = 0; i < H*16; i++) {
            for(int j = 0; j < W*16; j++) {
                float sum = 0;
                for(int k = 0; k < K*16; k++) {
                    sum += i_ref[k*16*H + i]*i_ref[256*K*H + j*K*16+k];
                }
                o_ref[i*W*16+j] = sum;
            }
        }
    }
    template<int H, int W, int NW, typename _K, kittens::ducks::wgmma::normal L1, kittens::ducks::wgmma::normal L2>
    __device__ static void device_func(const kittens::bf16 *input, kittens::bf16 *output) {
        constexpr int K = _K::value;
        extern __shared__ kittens::alignment_dummy __shm[]; // this is the CUDA shared memory
        kittens::shared_allocator<1024> al((int*)&__shm[0]); 
        kittens::st_bf<K, H, L1> &a = al.allocate<kittens::st_bf<K, H, L1>>();
        kittens::st_bf<W, K, L2> &b = al.allocate<kittens::st_bf<W, K, L2>>();
        kittens::rt_fl<1, W> c;
        __shared__ cuda::barrier<cuda::thread_scope::thread_scope_block> barrier;
        if (threadIdx.x == 0) {init(&barrier, kittens::WARPGROUP_THREADS);}
        __syncthreads();
        kittens::warpgroup::load_async(a, input, H*16, barrier);
        kittens::warpgroup::load_async(b, input+a.num_elements, K*16, barrier);
        barrier.arrive_and_wait();
        kittens::warpgroup::mma_fence(c);
        kittens::warpgroup::mm_AtBt(c, a, b);
        kittens::warpgroup::mma_commit_group();
        kittens::warpgroup::mma_async_wait();
        kittens::warpgroup::store(output, c, W*16);
    }
};

// Due to the strange sizes instantiated, we need a custom base wrapper here
template<typename test, int H, int W, int NUM_WORKERS, typename _K, typename... args>
struct mma_wrapper_2d {
    static void run(test_data& results) {
        using namespace kittens;
        constexpr int K = _K::value;
        test_info this_result;
        this_result.label = generate_test_name<H,W,NUM_WORKERS,_K,args...>(test::test_identifier);
        if constexpr (test::template valid<H, W, NUM_WORKERS, _K, args...>::value) {
            // initialize
            bf16 *d_i, *d_o;
            std::vector<float> i_ref((H+W)*K*256);
            std::vector<float> o_ref(H*W*256);
            initialize(&d_i, &d_o, i_ref, o_ref);
            // run kernel
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                global_wrapper_2d<test), H, W, NUM_WORKERS, _K, args...>,
                hipFuncAttributeMaxDynamicSharedMemorySize,
                kittens::MAX_SHARED_MEMORY
            );
            global_wrapper_2d<test, H, W, NUM_WORKERS, _K, args...><<<1, NUM_WORKERS*32, kittens::MAX_SHARED_MEMORY>>>(d_i, d_o);
            // fill in correct results on cpu
            test::template host_func<H, W, NUM_WORKERS, _K, args...>(i_ref, o_ref);
            // check and cleanup
            this_result.result = validate(d_i, d_o, i_ref, o_ref, this_result.label, W*16, 0.02); // wgmma's sometimes produce small errors. this appears to be hardware.
        }
        else {
            this_result.result = test_result::INVALID;
        }
        results.push_back(this_result);
    }
};
template<typename test, int H, int MAX_W, int NUM_WORKERS=1, typename... args> using mma_sweep_width = loop_w<mma_wrapper_2d, test, H, MAX_W, NUM_WORKERS, H, MAX_W, args...>;
template<typename test, int MAX_W, typename... args> using mma_sweep_width_warpgroup = mma_sweep_width<test, 4, MAX_W, 4, args...>;

using namespace kittens::ducks::st_layout;
// If 1 and 3 work, the others likely will too.
using I1_t = std::integral_constant<int, 1>;
using I2_t = std::integral_constant<int, 2>;
using I3_t = std::integral_constant<int, 3>;
using I4_t = std::integral_constant<int, 4>;
using I5_t = std::integral_constant<int, 5>;
using I6_t = std::integral_constant<int, 6>;
using I7_t = std::integral_constant<int, 7>;
using I8_t = std::integral_constant<int, 8>;
void group::wgmma::mma::tests(test_data &results) {
    std::cout << "\n ----- Starting ops/warpgroup/wgmma/mma tests! -----\n" << std::endl;
    constexpr int SIZE = INTENSITY_1 ? 1 :
                         INTENSITY_2 ? 2 : 
                         INTENSITY_3 ? 4 :
                         INTENSITY_4 ? 8 : -1;
    mma_sweep_width_warpgroup<test_mma_AB,   SIZE, I1_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_ABt,  SIZE, I1_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_AtB,  SIZE, I1_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_AtBt, SIZE, I1_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_AB,   SIZE, I2_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_ABt,  SIZE, I2_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_AtB,  SIZE, I2_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_AtBt, SIZE, I2_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_AB,   SIZE, I3_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_ABt,  SIZE, I3_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_AtB,  SIZE, I3_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_AtBt, SIZE, I3_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_AB,   SIZE, I4_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_ABt,  SIZE, I4_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_AtB,  SIZE, I4_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_AtBt, SIZE, I4_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_AB,   SIZE, I5_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_ABt,  SIZE, I5_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_AtB,  SIZE, I5_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_AtBt, SIZE, I5_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_AB,   SIZE, I6_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_ABt,  SIZE, I6_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_AtB,  SIZE, I6_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_AtBt, SIZE, I6_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_AB,   SIZE, I7_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_ABt,  SIZE, I7_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_AtB,  SIZE, I7_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_AtBt, SIZE, I7_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_AB,   SIZE, I8_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_ABt,  SIZE, I8_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_AtB,  SIZE, I8_t, wgmma_interleave, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_AtBt, SIZE, I8_t, wgmma_interleave, wgmma_interleave>::run(results);

    mma_sweep_width_warpgroup<test_mma_AB,   SIZE, I1_t, wgmma_swizzle, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_ABt,  SIZE, I1_t, wgmma_swizzle, wgmma_swizzle>::run(results);
    mma_sweep_width_warpgroup<test_mma_AB,   SIZE, I2_t, wgmma_swizzle, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_ABt,  SIZE, I2_t, wgmma_swizzle, wgmma_swizzle>::run(results);
    mma_sweep_width_warpgroup<test_mma_AB,   SIZE, I3_t, wgmma_swizzle, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_ABt,  SIZE, I3_t, wgmma_swizzle, wgmma_swizzle>::run(results);
    mma_sweep_width_warpgroup<test_mma_AB,   SIZE, I4_t, wgmma_swizzle, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_ABt,  SIZE, I4_t, wgmma_swizzle, wgmma_swizzle>::run(results);
    mma_sweep_width_warpgroup<test_mma_AB,   SIZE, I5_t, wgmma_swizzle, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_ABt,  SIZE, I5_t, wgmma_swizzle, wgmma_swizzle>::run(results);
    mma_sweep_width_warpgroup<test_mma_AB,   SIZE, I6_t, wgmma_swizzle, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_ABt,  SIZE, I6_t, wgmma_swizzle, wgmma_swizzle>::run(results);
    mma_sweep_width_warpgroup<test_mma_AB,   SIZE, I7_t, wgmma_swizzle, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_ABt,  SIZE, I7_t, wgmma_swizzle, wgmma_swizzle>::run(results);
    mma_sweep_width_warpgroup<test_mma_AB,   SIZE, I8_t, wgmma_swizzle, wgmma_interleave>::run(results);
    mma_sweep_width_warpgroup<test_mma_ABt,  SIZE, I8_t, wgmma_swizzle, wgmma_swizzle>::run(results);
}

#endif
