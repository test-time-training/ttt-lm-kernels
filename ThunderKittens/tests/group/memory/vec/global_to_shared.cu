#include "hip/hip_runtime.h"
#include "global_to_shared.cuh"

#ifdef TEST_GROUP_MEMORY_VEC_GLOBAL_TO_SHARED

struct vec_load_store {
    template<int S, int NW> using valid = std::bool_constant<S%NW==0 && S<=64>;
    static inline const std::string test_identifier = "shared_vec_loadstore";
    template<int S, int NW> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        o_ref = i_ref; // overwrite the whole thing
    }
    template<int S, int NW> __device__ static void device_func(const kittens::bf16 *input, kittens::bf16 *output) {
        using G = kittens::group<NW>;
        extern __shared__ kittens::alignment_dummy __shm[]; // this is the CUDA shared memory
        kittens::shared_allocator<16> al((int*)&__shm[0]); 
        kittens::col_vec<kittens::st_bf<S, S>> &shared_vec = al.allocate<kittens::col_vec<kittens::st_bf<S, S>>>();
        G::load(shared_vec, input);
        G::store(output, shared_vec);
    }
};

void group::memory::vec::global_to_shared::tests(test_data &results) {
    std::cout << "\n ----- Starting ops/group/memory/vec/global_to_shared tests! -----\n" << std::endl;
    constexpr int SIZE = INTENSITY_1 ? 2  :
                         INTENSITY_2 ? 4  : 
                         INTENSITY_3 ? 8  :
                         INTENSITY_4 ? 16 : -1;
                         
    sweep_size_1d<vec_load_store, SIZE, 2>::run(results);
    sweep_size_1d<vec_load_store, SIZE, 4>::run(results);
    sweep_size_1d<vec_load_store, SIZE, 12>::run(results);
}

#endif