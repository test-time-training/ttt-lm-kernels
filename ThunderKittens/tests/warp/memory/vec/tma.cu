#include "hip/hip_runtime.h"
#include "tma.cuh"

#ifdef TEST_WARP_MEMORY_VEC_TMA

struct test_load { // load with TMA, write out normally
    template<int S, int NW> using valid = std::bool_constant<NW == 1 && S<=64 && S%4==0>; // S%4 ensures alignment
    static inline const std::string test_identifier = "tma_load_vec";
    template<int S, int NW> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        o_ref = i_ref; // overwrite the whole thing
    }
    template<int S, int NW>
    __device__ static void device_func(const kittens::bf16 *input, kittens::bf16 *output, CUtensorMap* tma_desc_input, CUtensorMap* tma_desc_output) {
        extern __shared__ kittens::alignment_dummy __shm[]; // this is the CUDA shared memory
        kittens::tma_allocator al((int*)&__shm[0]); 
        kittens::row_vec<kittens::st_bf<S, S>> (&shared_vec)[4] = al.allocate<kittens::row_vec<kittens::st_bf<S, S>>, 4>();
        
        __shared__ kittens::tma::barrier smem_barrier; 
        kittens::tma::init_barrier<typeof(shared_vec[0]), 4>(smem_barrier);
        for(int i = 0; i < 4; i++) {
            kittens::tma::load_async(shared_vec[i], tma_desc_input, smem_barrier, i);
        }
        kittens::tma::arrive_and_wait(smem_barrier, 0);
        kittens::store(output, shared_vec[0]);
        kittens::store(output + shared_vec[0].length, shared_vec[1]);
        kittens::store(output + 2*shared_vec[0].length, shared_vec[2]);
        kittens::store(output + 3*shared_vec[0].length, shared_vec[3]);
    }
};
struct test_store { // load normally, store with TMA
    template<int S, int NW> using valid = std::bool_constant<NW == 1 && S<=64 && S%4==0>; // S%4 ensures alignment
    static inline const std::string test_identifier = "tma_store_vec";
    template<int S, int NW> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        o_ref = i_ref; // overwrite the whole thing
    }
    template<int S, int NW>
    __device__ static void device_func(const kittens::bf16 *input, kittens::bf16 *output, CUtensorMap* tma_desc_input, CUtensorMap* tma_desc_output) {
        extern __shared__ kittens::alignment_dummy __shm[]; // this is the CUDA shared memory
        kittens::tma_allocator al((int*)&__shm[0]); 
        kittens::row_vec<kittens::st_bf<S, S>> (&shared_vec)[4] = al.allocate<kittens::row_vec<kittens::st_bf<S, S>>, 4>();
        
        kittens::load(shared_vec[0], input);
        kittens::load(shared_vec[1], input + shared_vec[0].length);
        kittens::load(shared_vec[2], input + 2*shared_vec[0].length);
        kittens::load(shared_vec[3], input + 3*shared_vec[0].length);
        __syncwarp();
        for(int i = 0; i < 4; i++) {
            kittens::tma::store_async(tma_desc_output, shared_vec[i], i);
        }
        kittens::tma::store_commit_group();
        kittens::tma::store_async_wait<0>();
    }
};
struct test_store_add_reduce {
    template<int S, int NW> using valid = std::bool_constant<NW == 1 && S<=64 && S%4==0>; // S%4 ensures alignment
    static inline const std::string test_identifier = "tma_store_add_reduce_vec";
    template<int S, int NW> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        // i_ref is reduced onto output
        for (int i = 0; i < o_ref.size(); i++) {
            o_ref[i] = i_ref[i] + i_ref[i]; 
        }
    }
    template<int S, int NW>
    __device__ static void device_func(const kittens::bf16 *input, kittens::bf16 *output, CUtensorMap* tma_desc_input, CUtensorMap* tma_desc_output) {
        extern __shared__ kittens::alignment_dummy __shm[]; // this is the CUDA shared memory
        kittens::tma_allocator al((int*)&__shm[0]); 
        kittens::row_vec<kittens::st_bf<S, S>> (&shared_vec)[4] = al.allocate<kittens::row_vec<kittens::st_bf<S, S>>, 4>();
        
        kittens::load(shared_vec[0], input);
        kittens::load(shared_vec[1], input + shared_vec[0].length);
        kittens::load(shared_vec[2], input + 2*shared_vec[0].length);
        kittens::load(shared_vec[3], input + 3*shared_vec[0].length);
        __syncwarp();
        for(int i = 0; i < 4; i++) {
            kittens::tma::store_add_async(tma_desc_output, shared_vec[i], i);
        }
        kittens::tma::store_commit_group();
        for(int i = 0; i < 4; i++) {
            kittens::tma::store_add_async(tma_desc_output, shared_vec[i], i);
        }
        kittens::tma::store_commit_group();
        kittens::tma::store_async_wait<0>();
    }
};
struct test_store_min_reduce {
    template<int S, int NW> using valid = std::bool_constant<NW == 1 && S<=64 && S%4==0>; // S%4 ensures alignment
    static inline const std::string test_identifier = "tma_store_min_reduce_vec";
    template<int S, int NW> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        // i_ref is reduced onto output
        for (int i = 0; i < o_ref.size(); i++) {
            o_ref[i] = std::min(i_ref[i], i_ref[i]);
        }
    }
    template<int S, int NW>
    __device__ static void device_func(const kittens::bf16 *input, kittens::bf16 *output, CUtensorMap* tma_desc_input, CUtensorMap* tma_desc_output) {
        extern __shared__ kittens::alignment_dummy __shm[]; // this is the CUDA shared memory
        kittens::tma_allocator al((int*)&__shm[0]); 
        kittens::row_vec<kittens::st_bf<S, S>> (&shared_vec)[4] = al.allocate<kittens::row_vec<kittens::st_bf<S, S>>, 4>();
        
        kittens::load(shared_vec[0], input);
        kittens::load(shared_vec[1], input + shared_vec[0].length);
        kittens::load(shared_vec[2], input + 2*shared_vec[0].length);
        kittens::load(shared_vec[3], input + 3*shared_vec[0].length);
        __syncwarp();
        for(int i = 0; i < 4; i++) {
            kittens::tma::store_add_async(tma_desc_output, shared_vec[i], i);
        }
        kittens::tma::store_commit_group();
        kittens::tma::store_async_wait<0>();
    }
};
struct test_store_max_reduce {
    template<int S, int NW> using valid = std::bool_constant<NW == 1 && S<=64 && S%4==0>; // S%4 ensures alignment
    static inline const std::string test_identifier = "tma_store_max_reduce_vec";
    template<int S, int NW> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        // i_ref is reduced onto output
        for (int i = 0; i < o_ref.size(); i++) {
            o_ref[i] = std::max(i_ref[i], i_ref[i]);
        }
    }
    template<int S, int NW>
    __device__ static void device_func(const kittens::bf16 *input, kittens::bf16 *output, CUtensorMap* tma_desc_input, CUtensorMap* tma_desc_output) {
        extern __shared__ kittens::alignment_dummy __shm[]; // this is the CUDA shared memory
        kittens::tma_allocator al((int*)&__shm[0]); 
        kittens::row_vec<kittens::st_bf<S, S>> (&shared_vec)[4] = al.allocate<kittens::row_vec<kittens::st_bf<S, S>>, 4>();
        
        kittens::load(shared_vec[0], input);
        kittens::load(shared_vec[1], input + shared_vec[0].length);
        kittens::load(shared_vec[2], input + 2*shared_vec[0].length);
        kittens::load(shared_vec[3], input + 3*shared_vec[0].length);
        __syncwarp();
        for(int i = 0; i < 4; i++) {
            kittens::tma::store_add_async(tma_desc_output, shared_vec[i], i);
        }
        kittens::tma::store_commit_group();
        kittens::tma::store_async_wait<0>();
    }
};

template<typename Ker, int S, int NW, typename... args>
static __global__ void tma_global_wrapper_1d(const kittens::bf16 *input, kittens::bf16 *output, CUtensorMap* tma_desc_input, CUtensorMap* tma_desc_output) {
    Ker::template device_func<S, NW, args...>(input, output, tma_desc_input, tma_desc_output);
}
template<typename test, int S, int NUM_WORKERS, typename... args>
struct tma_wrapper_1d {
    static void run(test_data& results) {
        test_info this_result;
        this_result.label = generate_test_name<S,NUM_WORKERS, args...>(test::test_identifier);
        if constexpr (test::template valid<S, NUM_WORKERS, args...>::value) {
            constexpr int SIZE = S*16 * 4; // 4 for additional TMA dimension
            // initialize
            kittens::bf16 *d_i, *d_o;
            std::vector<float> i_ref(SIZE);
            std::vector<float> o_ref(SIZE);
            initialize(&d_i, &d_o, i_ref, o_ref);
            // initialize TMA descriptors
            CUtensorMap *i_desc = kittens::tma::allocate_and_create_tensor_map<kittens::row_vec<kittens::st_bf<S, S>>>(d_i, 4);
            CUtensorMap *o_desc = kittens::tma::allocate_and_create_tensor_map<kittens::row_vec<kittens::st_bf<S, S>>>(d_o, 4);
            // run kernel
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                tma_global_wrapper_1d<test), S, NUM_WORKERS, args...>,
                hipFuncAttributeMaxDynamicSharedMemorySize,
                kittens::MAX_SHARED_MEMORY
            );
            tma_global_wrapper_1d<test, S, NUM_WORKERS, args...><<<1, NUM_WORKERS*32, kittens::MAX_SHARED_MEMORY>>>(d_i, d_o, i_desc, o_desc);
            // fill in correct results on cpu
            test::template host_func<S, NUM_WORKERS, args...>(i_ref, o_ref);
            // check and cleanup
            this_result.result = validate(d_i, d_o, i_ref, o_ref, this_result.label, S*16);
            hipFree(i_desc);
            hipFree(o_desc);
        }
        else {
            this_result.result = test_result::INVALID;
        }
        results.push_back(this_result);
    }
};
template<typename test, int MAX_S=8, typename... args>
using tma_sweep_size_1d_warp = loop_s<tma_wrapper_1d, test, MAX_S, 1, MAX_S, args...>;

void warp::memory::vec::tma::tests(test_data &results) {
    std::cout << "\n ----- Starting ops/warp/memory/vec/tma tests! -----\n" << std::endl;
    constexpr int SIZE = INTENSITY_1 ? 2  :
                         INTENSITY_2 ? 4  : 
                         INTENSITY_3 ? 8  :
                         INTENSITY_4 ? 16 : -1;

    tma_sweep_size_1d_warp<test_load,             SIZE>::run(results);
    tma_sweep_size_1d_warp<test_store,            SIZE>::run(results);
    tma_sweep_size_1d_warp<test_store_add_reduce, SIZE>::run(results);
    tma_sweep_size_1d_warp<test_store_min_reduce, SIZE>::run(results);
    tma_sweep_size_1d_warp<test_store_max_reduce, SIZE>::run(results);
}

#endif