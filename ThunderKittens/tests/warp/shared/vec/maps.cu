#include "hip/hip_runtime.h"
#include "maps.cuh"

#ifdef TEST_WARP_SHARED_VEC_MAPS

struct vec_add1 {
    template<int S, int NW>
    using valid = std::bool_constant<NW == 1 && S<=64>; // this is warp-level
    static inline const std::string test_identifier = "shared_vec_add1";
    template<int S, int NW>
    __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        for(int i = 0; i < o_ref.size(); i++) o_ref[i] = i_ref[i]+1.; // overwrite the whole thing
    }
    template<int S, int NW>
    __device__ static void device_func(const kittens::bf16 *input, kittens::bf16 *output) {
        __shared__ kittens::col_vec<kittens::st_bf<S, S>> vec;
        kittens::load(vec, input);
        kittens::add(vec, vec, __float2bfloat16(1.));
        kittens::store(output, vec);
    }
};

void warp::shared::vec::maps::tests(test_data &results) {
    std::cout << "\n ----- Starting ops/warp/shared/vec/maps tests! -----\n" << std::endl;
    constexpr int SIZE = INTENSITY_1 ? 2  :
                         INTENSITY_2 ? 4  : 
                         INTENSITY_3 ? 8  :
                         INTENSITY_4 ? 16 : -1;
                         
    sweep_size_1d_warp<vec_add1, SIZE>::run(results);
}

#endif